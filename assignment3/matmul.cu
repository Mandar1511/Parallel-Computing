#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>
#include <cmath>
#include <ctime>
#include <chrono>
using namespace std;

__global__ void matrixMul(float *devA, float *devB, float *devC, int n) {
    int i = blockIdx.y * blockDim.y + threadIdx.y; // row
    int j = blockIdx.x * blockDim.x + threadIdx.x; // col
    float element = 0.0f;
    if (i < n && j < n) {
        for (int k = 0; k < n; k++) {
            // dot product
            element += devA[i * n + k] * devB[k * n + j];
        }
        devC[i * n + j] = element;
    }
}

void initializeMatrix(float *matrix, int n) {
    for (int i = 0; i < n * n; ++i) {
        matrix[i] = -1.0f + static_cast<float>(rand()) / (static_cast<float>(RAND_MAX / 2.0f));
    }
}

void validate(float *hostC, float *hostA, float *hostB, int n) {
    auto now = chrono::system_clock::now();
    float *cpu_result = new float[n * n];
    // compute matrix multiplication on host
    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < n; ++j) {
            float element = 0.0f;
            for (int k = 0; k < n; ++k) {
                element += hostA[i * n + k] * hostB[k * n + j];
            }
            cpu_result[i * n + j] = element;
        }
    }
    cout << "Host Computation time in ms: " << std::chrono::duration_cast<std::chrono::milliseconds>(chrono::system_clock::now() - now).count() << "\n\n";

    bool isCorrect = true;
    const float threshold = 2 * (1e-2);
    for (int i = 0; i < n * n; ++i) {
        if (fabs(hostC[i] - cpu_result[i]) > threshold) {
            isCorrect = false;
            cout << "Mismatch at index " << i << " "<<hostC[i] << " "<<cpu_result[i]<<endl;
        }
    }
    if (isCorrect) {
        cout << "Results are correct." << endl;
    } else {
        cout << "Results are incorrect." << endl;
    }
    delete[] cpu_result;
}

int main(int argc, char* argv[]) {
    if (argc != 2) {
        cout << "provide matrix size" << endl;
        return 1;
    }
    int n = atoi(argv[1]);
    int size_in_bytes = n * n * sizeof(float);

    // Allocate host memory
    float *hostA = new float[size_in_bytes];
    float *hostB = new float[size_in_bytes];
    float *hostC = new float[size_in_bytes];

    // Initialize matrices in host memory
    initializeMatrix(hostA, n);
    initializeMatrix(hostB, n);
    for (int i = 0; i < n * n; ++i) {
        hostC[i] = 0.0f;
    }

    // Allocate device memory
    float *devA, *devB, *devC;
    hipMalloc(&devA, size_in_bytes);
    hipMalloc(&devB, size_in_bytes);
    hipMalloc(&devC, size_in_bytes);
    auto now1 = chrono::system_clock::now();
    hipMemcpy(devA, hostA, size_in_bytes, hipMemcpyHostToDevice);
    hipMemcpy(devB, hostB, size_in_bytes, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(32, 32);
    dim3 blocksPerGrid(ceil(n/32.0), ceil(n/32.0));

    auto now = chrono::system_clock::now();
    // Lauch kernel function
    matrixMul<<<blocksPerGrid, threadsPerBlock>>>(devA, devB, devC, n);
    hipDeviceSynchronize();
    cout << "Device Computation time in ms: " << std::chrono::duration_cast<std::chrono::milliseconds>(chrono::system_clock::now() - now).count() << "\n\n";

    // Copy data from device to host
    hipMemcpy(hostC, devC, size_in_bytes, hipMemcpyDeviceToHost);
    cout << "Device Computation time including data transfer in ms: " << std::chrono::duration_cast<std::chrono::milliseconds>(chrono::system_clock::now() - now1).count() << "\n\n";

    // Validate results
    validate(hostC, hostA, hostB, n);
    // free memory on host
    delete[] hostA;
    delete[] hostB;
    delete[] hostC;
    // free memory on device
    hipFree(devA);
    hipFree(devB);
    hipFree(devC);
    hipDeviceReset();

    return 0;
}
